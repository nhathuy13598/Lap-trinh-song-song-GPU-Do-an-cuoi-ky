#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#define LIBRARY 0
#define HOST 1
#define DEVICE 2
#define CHECK(call)                                            \
{                                                              \
	const hipError_t error = call;                            \
	if (error != hipSuccess)                                  \
	{                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
		fprintf(stderr, "code: %d, reason: %s\n", error,       \
				hipGetErrorString(error));                    \
		exit(1);                                               \
	}                                                          \
}

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
		hipEventSynchronize(start);
	}

	void Stop() { hipEventRecord(stop, 0); }

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

// Sequential radix sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Ta sẽ sử dụng ý tưởng từ thuật toán sắp xếp tuần tự mới chứ không phải
// thuật toán mà ta đã sử dụng trong bài tập số 3
/**
    Sắp xếp tuần tự trên host
    @blockSize kích thước một block mà ta sẽ duyệt (Ta vẫn duyệt tuần tự)
*/
void sortByHost(const uint32_t *in, int n, uint32_t *out, int nBits, int blockSize)
{
	int nBins = 1 << nBits; // Số lượng bin là 2^nBits

	uint32_t *src = (uint32_t *)malloc(n * sizeof(uint32_t));			   // Biến tạm để lưu trữ dữ liệu input
	memcpy(src, in, n * sizeof(uint32_t));		// Sao chép dữ liệu từ in vào src
	uint32_t *originalSrc = src;		   		// Use originalSrc to free memory later
	uint32_t *dst = out;				   		// Mảng kết quả

	// [TODO]: Khởi tạo các mảng cần thiết khi chạy
	int sizeHist = nBins * ((n - 1) / blockSize +1);	// Tính kích thước của mảng listLocalHist sizeHist = Số bin x Số lượng block
	int *listLocalHist = (int *)malloc(sizeHist * sizeof(int)); 			// Mảng chứa các localHist
	int *listLocalHistConvert = (int *)malloc(sizeHist * sizeof(int)); 		// Mảng chuyển đổi của listLocalHistConvert

	int *histScan = (int *)malloc(sizeHist * sizeof(int));		// Mảng exclusive scan của listLocalHistConvert

	int *eleBefore =(int *)malloc(n * sizeof(int)); 			// Mảng chứa chỉ số phân tử đứng trước và bằng nó trong từng block

	int numBlock = (n - 1) / blockSize + 1; // Số lượng các block cần thiết
	for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
	{
		// TODO: Mỗi block tính local histogram của digit-đang-xét trên phần
		// dữ liệu của mình và chép vào mảng listLocalHist
		memset(listLocalHist, 0, sizeHist * sizeof(int)); // Gán mảng listLocalHist bằng 0
		for (int blkIdx = 0; blkIdx < numBlock; blkIdx++)
		{
			int start = blkIdx * blockSize; 								// Chỉ số bắt đầu của block
			int end = (blkIdx == numBlock - 1)? n : start + blockSize; 		// Chỉ số kết thúc của block
			for (int index = start; index < end; index++)
			{ // Duyệt tất cả phần tử của block
				int bin = (src[index] >> bit) & (nBins - 1);
				listLocalHist[blkIdx * nBins + bin]++;
			}
		}

		// [DEBUG]: In ra mảng listLocalHist
		/*printf("Mang listLocalHist: ");
    	for(int i=0; i < sizeHist; i++){
        	printf("%d ", listLocalHist[i]);
    	}
    	printf("\n");*/

		// TODO: Với mảng 2 chiều mà mỗi dòng là local hist của một block,
		// thực hiện exclusive scan trên mảng một chiều gồm các cột
		// nối lại với nhau (Xem slide để hiểu rõ)
		int indexLLHC = 0; // Chỉ số trong mảng listLocalHistConvert
		for (int i = 0; i < nBins; i++)
		{ // Duyệt tất cả các phần tử trong một localHist
			for (int j = 0; j < numBlock; j++)
			{ // Duyệt tất cả các localHist
				listLocalHistConvert[indexLLHC++] =listLocalHist[i + j * nBins]; 	// i là chỉ số bin trong localHist
												  									// j * nBins là chỉ số của block
			}
		}
		// [DEBUG]: In ra mảng listLocalHistConvert
		/*printf("Mang listLocalHistConvert: ");
		for(int i=0; i<sizeHist; i++){
			printf("%d ", listLocalHistConvert[i]);
		}
    	printf("\n");*/

		// Tính histScan (exculusive scan) cho mảng listLocalHistConvert
		histScan[0] = 0;
		for (int i = 1; i < sizeHist; i++)
		{
			histScan[i] = histScan[i - 1] + listLocalHistConvert[i - 1];
		}

		// [DEBUG]: In ra mảng histScan
		/*printf("Mang histScan: ");
		for(int i=0; i<sizeHist; i++){
			printf("%d ", histScan[i]);
		}
		printf("\n");*/

		// TODO: Mỗi block thực hiện scatter phần dữ liệu của mình xuống
		// mảng output dựa vào kết quả scan ở trên
		//      ▪ Mỗi block sắp xếp cục bộ phần dữ liệu của mình theo digit đang
		//          xét (dùng Radix Sort với k=1 bit và làm trên SMEM)
		//      ▪ Mỗi block tính chỉ số bắt đầu (xét cục bộ trong block) của mỗi giá
		//          trị digit
		//      ▪ Mỗi thread trong block tính số lượng phần tử đứng trước mình
		//          trong block có digit-đang-xét bằng digit-đang-xét của phần tử mà
		//          mình phụ trách
		//      ▪ Mỗi thread trong block tính rank và thực hiện scatter

		// Sắp xếp các data trong block tăng dần theo Bubble Sort
		// Ta sẽ thực hiện luôn trên mảng src để tiết kiệm bộ nhớ
		for (int blkIdx = 0; blkIdx < numBlock; blkIdx++)
		{	// Duyệt từng block
			int start = blkIdx * blockSize; 								// Chỉ số bắt đầu của block
			int end = (blkIdx == numBlock - 1) ? n : start + blockSize; 	// Chỉ số kết thúc của block
			for (int x = end - start; x >= 1; x--)
			{ // Ta tưởng tượng đây là sắp xếp mảng có end - start phần tử
				for (int y = 0; y < x - 1; y++)
				{
					int first = (src[blkIdx * blockSize + y] >> bit) & (nBins - 1);
					int second = (src[blkIdx * blockSize + y + 1] >> bit) & (nBins - 1);
					if (first > second)
					{
						uint32_t temp = src[blkIdx * blockSize + y];
						src[blkIdx * blockSize + y] = src[blkIdx * blockSize + y + 1];
						src[blkIdx * blockSize + y + 1] = temp;
					}
				}
			}
		}

		// [DEBUG]: In ra mảng sortBlockData
		/*printf("Mang da duoc sap xep theo block: ");
		for(int i=0; i<n; i++){
			printf("%d ", src[i]);
		}
		printf("\n");*/

		// Tính chỉ số bắt đầu trong block và tính luôn số lượng
		// phần tử giống nó và đứng trước nó
		memset(eleBefore, 0, n * sizeof(int)); // Khởi tạo mảng chứa các phần tử đứng trước bằng 0
		for (int blkIdx = 0; blkIdx < numBlock; blkIdx++)
		{
			int start = blkIdx * blockSize; // Chỉ số bắt đầu của block
			int end = (blkIdx == numBlock - 1) ? n : start + blockSize; // Chỉ số kết thúc của block
			for (int index = 1; index < end - start; index++)
			{
				int first = (src[blkIdx * blockSize + index - 1] >> bit) & (nBins - 1);
				int second = (src[blkIdx * blockSize + index] >> bit) & (nBins - 1);
				if (first == second)
				{
					eleBefore[blkIdx * blockSize + index] = eleBefore[blkIdx * blockSize + index - 1] + 1;
				}
			}
		}
		// [DEBUG]: In ra mảng eleBefore
		/*printf("Mang eleBefore: ");
		for(int index=0; index<n; index++){
			printf("%d ", eleBefore[index]);
		}
		printf("\n");*/

		// Tính rank và scatter
		for (int index = 0; index < n; index++)
		{
			int blIdx = index / blockSize;
			int bin = (src[index] >> bit) & (nBins - 1);
			int rank = histScan[bin * numBlock + blIdx] + eleBefore[index];
			dst[rank] = src[index];
		}

		// [DEBUG]: Mang dst
		/*printf("Mang dst: ");
		for (int index = 0; index < n; index++) {
			printf("%d ", dst[index]);
		}
		printf("\n");*/

		// TODO: Swap "src" and "dst"
		uint32_t *temp = src;
		src = dst;
		dst = temp;
	}
	// [DEBUG]: In mảng src
	/*printf("\nMang ket qua la: ");
	for(int index=0; index<n; index++){
		printf("%d ", src[index]);
	}
	printf("\n");*/

	// TODO: Copy result to "out"
	memcpy(out, src, n * sizeof(uint32_t));

	// Free memories
	free(listLocalHist);
	free(histScan);
	free(originalSrc);
	free(listLocalHistConvert);
	free(eleBefore);
}

void sortByLibrary(const uint32_t *in, int n, uint32_t *out, int nBits)
{
	// TODO
	thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}

#define getBin(num) (((num) >> (bit)) & ((nBins)-1))

__global__ void histogramKernel(uint32_t *in, int n, int *histArr, int nBits, int bit)
{
	extern __shared__ uint32_t s_in[];
	int nBins = 1 << nBits; // Số lượng bin
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Gán các bin của Hist là 0
	for (int stride = threadIdx.x; stride < nBins; stride += blockDim.x){ // Số lượng bin có thể lớn hơn số thread
		s_in[stride] = 0;
	}
	//s_in[nBins + threadIdx.x] = (idx < n) ? in[idx] : 0;
	__syncthreads();
	
	if (idx < n){
		atomicAdd(&s_in[getBin(in[idx])], 1);
		//atomicAdd(&s_in[getBin(s_in[nBins + threadIdx.x])], 1);
	}
	__syncthreads();

	for (int stride = threadIdx.x; stride < nBins; stride += blockDim.x){ // Số lượng bin có thể lớn hơn số thread
		histArr[blockIdx.x + gridDim.x * stride] = s_in[stride];
	}
}

__global__ void scanBlkKernel(int *in, int n, int *out, int *blkSums)
{
	extern __shared__ int temp[];

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	// load input into shared memory. this is exclusive scan, so shift right by
	// one and set first element to 0
	temp[threadIdx.x] = (threadIdx.x > 0) ? in[idx - 1] : 0;
	__syncthreads();

	if (idx >= n)
	{
		return;
	}

	out[idx] = temp[threadIdx.x];
	__syncthreads();

	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if (threadIdx.x >= stride)
		{
			out[idx] += temp[threadIdx.x - stride];
		}
		__syncthreads();
		temp[threadIdx.x] = out[idx];
		__syncthreads();
	}
	if (blkSums != NULL && (idx == n - 1 || threadIdx.x == blockDim.x - 1))
	{
		blkSums[blockIdx.x] = out[idx] + in[idx];
	}
}

__global__ void addBlkKernel(int *in, int n, int *blkSums)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (blockIdx.x == 0 || idx >= n)
	{
		return;
	}
	in[idx] = in[idx] + blkSums[blockIdx.x - 1];
}

__global__ void transposeKernel(int *in, int n, int width, int *out)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= n)
	{
		return;
	}
	int x = idx % width;
	int y = idx / width;
	out[x * (n / width) + y] = in[idx];
}

__global__ void scatterKernel(uint32_t *in, int n, uint32_t *out,
								int *scanHistogramArrayTranspose, 
								int nBits, int bit)
{
	/*
	Smem sẽ gồm có ? phần dữ liệu
		1. blockDim.x phần tử (dữ liệu input)
		2. phần tử dummy có 1 phần tử
		3. blockDim.x phần tử (Chuỗi nhị phân)
		4. 2 ^ nBits phần tử (chứa chỉ số bắt đầu)
		5. 2 ^ nBits phần tử (chứa scanHistogramArrayTranspose cho từng block)
	*/
	int nBins = 1 << nBits; // Số lượng bin
	int size = blockDim.x; //  Số lượng phần tử trong block
	if (blockIdx.x == gridDim.x - 1){
		size = n - (gridDim.x - 1) * blockDim.x;
	}
	// Load dữ liệu từ in vào smem
	extern __shared__ uint32_t tp[];
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	tp[threadIdx.x] = (idx < n) ? in[idx] : 0;
	// Load scanHistogramArrayTranspose vào smem
	// Số thread có thể ít hơn số bin
	int startHistArr = 2 * blockDim.x + 1 + nBins;
	// if (threadIdx.x < nBins){
	// 	tp[startHistArr + threadIdx.x] = scanHistogramArrayTranspose[blockIdx.x * nBins + threadIdx.x];
	// }
	for (int i = 0; i < nBins; i += blockDim.x){
		if (threadIdx.x + i < nBins){
			tp[startHistArr + threadIdx.x + i] = scanHistogramArrayTranspose[blockIdx.x * nBins + threadIdx.x + i];
		}
	}
	__syncthreads();
	// FIXME: Debug
	/*if (threadIdx.x == 0){
		printf("Mang scan Hist: \n");
		for (int i = 0; i < nBins; i++){
			printf("%d %d\n", tp[startHistArr + i], scanHistogramArrayTranspose[blockIdx.x * nBins + threadIdx.x]);
			if (i == nBins - 1)
				printf("\n");
		}
	}
	__syncthreads();*/
	// Lấy ra nBits (bit) của các phần tử trong block với chỉ số bit đầu tiên là bit

	// Sắp xếp các phần tử trong block bằng nBits (bit) này
	int startBitArr = blockDim.x + 1; // Chỉ số bắt đầu chuỗi nhị phân
	int startBitScan = blockDim.x; // Chỉ số bắt đầu của mảng scan-chuỗi-nhị-phân
	int nZeros = 0; // Số lượng số 0
	for (int i = 0; i < nBits; i++){

		// Lấy chuỗi bit
		if (threadIdx.x < size){
			uint32_t oneBit = (getBin(tp[threadIdx.x]) >> i) & 1;
			tp[startBitArr + threadIdx.x] = oneBit;
		}
		//__syncthreads();

		// FIXME: Debug
		/*if (threadIdx.x == 0){
			printf("Chuoi bit: ");
			for (int i = 0; i < size; i++){
				printf("%d ", tp[startBitArr + i]);
				if (i == size - 1)
					printf("\n");
			}
		}
		__syncthreads();*/

		// Set giá trị cho chuỗi bit scan
		tp[blockDim.x] = 0;
		__syncthreads();
		// Scan chuỗi bit
		for (int stride = 1; stride < size; stride *= 2) {
			int temp = 0;
			if (threadIdx.x >= stride && threadIdx.x < size) {
				temp = tp[startBitScan + threadIdx.x - stride];
			}
			__syncthreads();
			if (threadIdx.x >= stride && threadIdx.x < size) {
				tp[startBitScan + threadIdx.x] += temp;
			}
			__syncthreads();
		}
		//__syncthreads();
		
		// FIXME: Debug
		/*if (threadIdx.x == 0){
			printf("Chuoi bit scan: ");
			for (int j = 0; j < size; j++){
				printf("%d ", tp[startBitScan + j]);
				if (j == size - 1)
					printf("\n");
			}
		}
		__syncthreads();*/

		// Scatter
		nZeros = size - tp[startBitScan + size - 1] - tp[startBitArr + size - 1];
		
		// Lấy phần tử trong mảng ra lưu lại
		uint32_t ele;
		if (threadIdx.x < size){
			ele = tp[threadIdx.x];
		}
		__syncthreads();
		if (threadIdx.x < size){
			uint32_t oneBit = (getBin(ele) >> i) & 1;
			if (oneBit == 0){
				int rank = threadIdx.x - tp[startBitScan + threadIdx.x];
				tp[rank] = ele;
			}
			else{
				int rank = nZeros + tp[startBitScan + threadIdx.x];
				tp[rank] = ele;
			}
		}
		__syncthreads();
		
		// FIXME: Debug
		/*if (threadIdx.x == 0){
			printf("So luong zero la: %d\n",nZeros);
			printf("Chuoi bit sau khi sap xep: ");
			for (int j = 0; j < size; j++){
				printf("%d ", tp[j]);
				if (j == size - 1)
					printf("\n");
			}
		}
		__syncthreads(); return;*/
	}

	// FIXME: Debug
	/*if (threadIdx.x == 0){
		printf("Mang da sap xep: ");
		for (int i = 0; i < size; i++){
			printf("%d ", tp[i]);
			if (i == size - 1)
				printf("\n");
		}
	}
	__syncthreads();*/

	// Tính chỉ số bắt đầu của từng bộ nBits (bit) trong block
	int startArrIdx = 2 * blockDim.x + 1; // Chỉ số bắt đầu của mảng chứa chỉ-số-bắt-đầu-của-từng-bộ-nBits
	if (threadIdx.x == 0){
		int bin = getBin(tp[threadIdx.x]);
		tp[startArrIdx + bin] = 0;
	}
	else if (threadIdx.x < size){
		if (getBin(tp[threadIdx.x]) != getBin(tp[threadIdx.x - 1])){
			tp[startArrIdx + getBin(tp[threadIdx.x])] = threadIdx.x;
		}
	}
	__syncthreads();

	// FIXME: Debug
	/*if (threadIdx.x == 0){
		printf("Mang chi so bat dau: ");
		for (int i=0; i < nBins; i++){
			printf("%d ", tp[startArrIdx + i]);
			if (i == nBins - 1)
				printf("\n");
		}
	}
	__syncthreads();*/

	// Tính số phần tử đứng trước nó theo từng bộ nBits (bit) của các phần tử trong block
	int startArrEleBef = blockDim.x; // Chỉ số bắt đầu của mảng chứa số-phần-tử-đứng-trước-nó
	int bin = getBin(tp[threadIdx.x]);
	tp[startArrEleBef + threadIdx.x] = threadIdx.x - tp[startArrIdx + bin];
	//__syncthreads(); // !CHÚ Ý: Đoạn này chưa hiểu tại sao không bị lỗi

	// FIXME: Debug
	/*if (threadIdx.x == 0){
		printf("Mang so luong phan tu bat dau: ");
		for (int i = 0; i < size; i++){
			printf("%d ", tp[startArrEleBef + i]);
			if (i == size - 1)
				printf("\n");
		}
	}
	__syncthreads();*/

	// FIXME: Debug
	/*if (threadIdx.x == 0){
		printf("Mang scan hist: ");
		for(int k = 0; k < nBins; k++){
			printf("%d ", scanHistogramArrayTranspose[k]);
			if(k == nBins - 1)
				printf("\n");
		}
	}*/

	// Scatter
	if (threadIdx.x < size){
		//int rank = scanHistogramArrayTranspose[blockIdx.x * nBins + bin] + tp[startArrEleBef + threadIdx.x];
		int rank = tp[startHistArr + bin] + tp[startArrEleBef + threadIdx.x]; 
		out[rank] = tp[threadIdx.x];
	}

	// FIXME: Debug
	/*__syncthreads();
	if (threadIdx.x == 0){
		printf("Mang sau khi da sap xep: ");
		for (int i = 0; i < size; i++){
			printf("%d ", out[i]);
			if (i == size - 1)
				printf("\n");
		}
	}*/
}


void sortByDevice(const uint32_t *in, int n, uint32_t *out, int nBits, int *blockSizes)
{

	// initialize data
	int nBins = 1 << nBits; // 2 ^ nBits
	int gridSizeHist = (n - 1) / blockSizes[0] + 1;
	int gridSizeScan = (gridSizeHist * nBins - 1) / blockSizes[1] + 1;

	// Allocate data on device
	int in_size = n * sizeof(uint32_t);
	int out_size = in_size;
	uint32_t *d_src, *d_dst;
	//uint32_t *dst = (uint32_t*)malloc(out_size);
	CHECK(hipMalloc(&d_src, in_size));
	CHECK(hipMalloc(&d_dst, out_size));
	CHECK(hipMemcpy(d_src, in, in_size, hipMemcpyHostToDevice));

	// Allocate another data on device
	size_t histArr_size = gridSizeHist * nBins * sizeof(int);
	size_t size_blksum = gridSizeScan * sizeof(int);
	//int *histArr = (int *)malloc(histArr_size);
	//int *scanHistArr = (int *)malloc(histArr_size);
	int *blkSums = (int *)malloc(size_blksum);
	int *d_histArr, *d_scanHistArr, *d_blkSums;
	CHECK(hipMalloc(&d_histArr, histArr_size));
	CHECK(hipMalloc(&d_scanHistArr, histArr_size));
	CHECK(hipMalloc(&d_blkSums, size_blksum));
	

	// Set time
	GpuTimer timer;
	float histTime, scanTime, addTime, transposeTime, scatterTime;
	histTime = scanTime = addTime = transposeTime = scatterTime = 0;
	for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
	{
		CHECK(hipMemset(d_histArr, 0, histArr_size));

		// TODO: Do histogram
		timer.Start();
		histogramKernel<<<gridSizeHist, blockSizes[0], 
							(/*blockSizes[0] +*/ nBins) * sizeof(uint32_t)>>>(d_src, n, d_histArr, nBits, bit);
		CHECK(hipGetLastError());
		timer.Stop();
		histTime += timer.Elapsed();

		// FIXME: Debug
		/*CHECK(hipMemcpy(histArr, d_histArr, histArr_size, hipMemcpyDeviceToHost));
		for (int i = 0; i < gridSizeHist * nBins; ++i) {
		  	printf("%d\t", histArr[i]);
			if (i == gridSizeHist * nBins - 1)
				printf("\n===|===|===|===|===|===|===|===|===|===\n");
		}*/
		

		// TODO: Scan histogram
		timer.Start();
		scanBlkKernel<<<gridSizeScan, blockSizes[1], blockSizes[1] * sizeof(int)>>>(d_histArr, gridSizeHist * nBins, d_scanHistArr, d_blkSums);
		CHECK(hipGetLastError());

		// copy result to host
		CHECK(hipMemcpy(blkSums, d_blkSums, size_blksum, hipMemcpyDeviceToHost));

		// scan vector blkSums
		for (int i = 1; i < gridSizeScan; ++i)
		{
			blkSums[i] += blkSums[i - 1];
		}
		timer.Stop();
		scanTime += timer.Elapsed();

		// copy data to device
		CHECK(hipMemcpy(d_blkSums, blkSums, size_blksum, hipMemcpyHostToDevice));

		// TODO: Add after scan
		timer.Start();
		addBlkKernel<<<gridSizeScan, blockSizes[1]>>>(d_scanHistArr, gridSizeHist * nBins, d_blkSums);
		CHECK(hipGetLastError());
		timer.Stop();
		addTime += timer.Elapsed();

		// FIXME: Debug
		/*CHECK(hipMemcpy(scanHistArr, d_scanHistArr, histArr_size, hipMemcpyDeviceToHost));
		for (int i = 0; i < gridSizeHist * nBins; ++i) {
		  	printf("%d\t", scanHistArr[i]);
		  	if(i == gridSizeHist * nBins - 1)
		  		printf("\n===|===|===|===|===|===|===|===|===|===\n");
		}*/
		

		// TODO: Transpose
		timer.Start();
		transposeKernel<<<gridSizeScan, blockSizes[1]>>>(d_scanHistArr, gridSizeHist * nBins, gridSizeHist, d_histArr);
		CHECK(hipGetLastError());
		timer.Stop();
		transposeTime += timer.Elapsed();

		// FIXME: Debug
		/*CHECK(hipMemcpy(scanHistArr, d_scanHistArrTranpose, histArr_size, hipMemcpyDeviceToHost));
		for (int i = 0; i < gridSizeHist * nBins; ++i) {
		  	printf("%d ", scanHistArr[i]);
			if(i == gridSizeHist * nBins - 1)
				printf("\n===|===|===|===|===|===|===|===|===|===\n");
		}*/
		

		// TODO: Scatter
		timer.Start();
		scatterKernel<<<gridSizeHist, blockSizes[0], 
						(2 * blockSizes[0] + 1 + 2 * nBins)* sizeof(uint32_t)>>>(d_src, n, d_dst, d_histArr, nBits, bit);
		CHECK(hipGetLastError());
		timer.Stop();
		scatterTime += timer.Elapsed();

		// FIXME: Debug
		/*CHECK(hipMemcpy(dst, d_dst, out_size, hipMemcpyDeviceToHost));
		for (int i = 0; i < n; ++i) {
		  	printf("%d\t", dst[i]);
			if (i == n - 1)
				printf("\n===|===|===|===|===|===|===|===|===|===\n");
		}*/
		//break;
		// Swap "src" and "dst"
		uint32_t *tp = d_src;
		d_src = d_dst;
		d_dst = tp;
	}
	// Print runtime
	printf("Hist Time: %.3f\n", histTime);
	printf("Scan Time: %.3f\n", scanTime);
	printf("Add Time: %.3f\n", addTime);
	printf("Transpose Time: %.3f\n", transposeTime);
	printf("Scatter Time: %.3f\n", scatterTime);

	// DONE: Copy result from "d_src" to "out"
	CHECK(hipMemcpy(out, d_src, n * sizeof(uint32_t), hipMemcpyDeviceToHost));

	// free memories
	CHECK(hipFree(d_src));
	CHECK(hipFree(d_dst));
	CHECK(hipFree(d_histArr));
	CHECK(hipFree(d_scanHistArr));
	CHECK(hipFree(d_blkSums))
	free(blkSums);
	//free(histArr);
	//free(scanHistArr);
	//free(dst);
}

// Radix sort
/*
        @type 	0 Sử dụng thư viện
                1 Sử dụng Host
                2 Sử dụng Device
*/
void sort(const uint32_t *in, int n, uint32_t *out, int nBits, int type,
		  int *blockSizes = NULL)
{
	GpuTimer timer;
	timer.Start();
	if (type == 0)
	{
		printf("\nRadix sort by library\n");
		sortByLibrary(in, n, out, nBits);
	}
	else if (type == 1)
	{
		printf("\nRadix sort by host\n");
		//sortByHost(in, n, out, nBits, 32);
	}
	else // use device
	{
		printf("\nRadix sort by device\n");
		sortByDevice(in, n, out, nBits, blockSizes);
	}
	timer.Stop();
	printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
	CHECK(hipGetDeviceProperties(&devProv, 0));
	printf("**********GPU info**********\n");
	printf("Name: %s\n", devProv.name);
	printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
	printf("Num SMs: %d\n", devProv.multiProcessorCount);
	printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
	printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
	printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
	printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
	printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
	printf("****************************\n");
}

void checkCorrectness(uint32_t *out, uint32_t *correctOut, int n)
{
	for (int i = 0; i < n; i++)
	{
		if (out[i] != correctOut[i])
		{
			printf("INCORRECT :(\n");
			return;
		}
	}
	printf("CORRECT :)\n");
}

void printArray(uint32_t *a, int n)
{
	for (int i = 0; i < n; i++)
		printf("%i ", a[i]);
	printf("\n");
}

int main(int argc, char **argv)
{
	// PRINT OUT DEVICE INFO
	printDeviceInfo();

	// SET UP INPUT SIZE
	int n = (1 << 24) + 1;
	//n = 1000000;
	//n = 10;
	printf("\nInput size: %d\n", n);

	// ALLOCATE MEMORIES
	size_t bytes = n * sizeof(uint32_t);
	uint32_t *in = (uint32_t *)malloc(bytes);
	uint32_t *out = (uint32_t *)malloc(bytes);		  // Device result
	uint32_t *correctOut = (uint32_t *)malloc(bytes); // Host result

	// SET UP INPUT DATA
	for (int i = 0; i < n; i++)
		in[i] = rand();
	// in[i] = rand() % 8;
	//uint32_t temp[10] = {3,2,5,7,9,9,8,8,1,1}; 
	//memcpy(in, temp, n * sizeof(uint32_t)); 
	//printArray(in, n);

	// SET UP NBITS
	int nBits = 4; // Default
	//nBits = 1;
	if (argc > 1)
		nBits = atoi(argv[1]);
	printf("\nNum bits per digit: %d\n", nBits);

	// DETERMINE BLOCK SIZES
	int blockSizes[2] = {512, 512}; // One for histogram, one for scan
	// hipDeviceProp_t devProv;
	// CHECK(hipGetDeviceProperties(&devProv, 0));
	// if (devProv.major <= 3 && devProv.minor <= 7)
	// {
	// 	blockSizes[0] = blockSizes[1] = devProv.maxThreadsPerMultiProcessor / 16;
	// }
	// else if (devProv.major <= 7 && devProv.minor <= 3)
	// {
	// 	blockSizes[0] = blockSizes[1] = devProv.maxThreadsPerMultiProcessor / 32;
	// }
	// else {
	// 	blockSizes[0] = blockSizes[1] = devProv.maxThreadsPerMultiProcessor / 16;
	// }
	if (argc == 4)
	{
		blockSizes[0] = atoi(argv[2]);
		blockSizes[1] = atoi(argv[3]);
	}
	printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0],
		   blockSizes[1]);

	// SORT BY LIBRARY
	sort(in, n, correctOut, nBits, LIBRARY);

	// SORT BY HOST
	sort(in, n, out, nBits, HOST);
	checkCorrectness(out, correctOut, n);

	// SORT BY DEVICE
	out[0] = 1; // Sửa lại mảng out để output sẽ khác
	sort(in, n, out, nBits, DEVICE, blockSizes);
	checkCorrectness(out, correctOut, n);

	// FREE MEMORIES
	free(in);
	free(out);
	free(correctOut);

	return EXIT_SUCCESS;
}
